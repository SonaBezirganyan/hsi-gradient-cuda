
#include <hip/hip_runtime.h>
#include <cstdint>

#define BLOCK_X_2D 16
#define BLOCK_Y_2D 16

__global__ void gm_morph_max_kernel(int NX, int NY, int NC, std::uint16_t* d_image, std::uint16_t* d_gm) {
    int i, j, ind;
    int ni, nj, nind;
    std::uint16_t max_val, min_val, nval;
    std::uint16_t max_marginal = 0;

    int off_x[] = {-1, 0, 1, -1, 1, -1, 0, 1};
    int off_y[] = {-1, -1, -1, 0, 0, 1, 1, 1};

    // define global indices
    i = threadIdx.x + blockIdx.x * blockDim.x;  
    j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < NX && j < NY) { // && i >= 0 && j >= 0
        ind = i + j * NX;
        for (int c = 0; c < NC; c++) {
            max_val = min_val = d_image[ind + c * NX * NY];
            for (int k = 0; k < 8; k++) {
                ni = i + off_x[k];
                nj = j + off_y[k];

                if (ni >= 0 && ni < NX && nj >= 0 && nj < NY) {
                    nind = ni + nj * NX;
                    nval = d_image[nind + c * NX * NY];
                    if (nval > max_val) {
                        max_val = nval;
                    } else if (nval < min_val) {
                        min_val = nval;
                    }
                }
            }
            if (max_val - min_val > max_marginal) {
                max_marginal = max_val - min_val;
            }
        }
        d_gm[ind] = max_marginal;
    }
}


void gm_morph_max(int NX, int NY, int NC, std::uint16_t* h_image, std::uint16_t* h_gm) {
    // initialisations
    int bx, by;
    std::uint16_t* d_image;
    std::uint16_t* d_gm;

    // allocate memory for arrays
    hipMalloc((void**) &d_image, NX * NY * NC * sizeof(std::uint16_t));
    hipMalloc((void**) &d_gm, NX * NY * sizeof(std::uint16_t));

    // data transfer CPU to GPU and initialisation
    hipMemcpy(d_image, h_image, NX * NY * NC * sizeof(std::uint16_t), hipMemcpyHostToDevice);

    // GPU processing
    bx = 1 + (NX - 1) / BLOCK_X_2D;
    by = 1 + (NY - 1) / BLOCK_Y_2D;

    dim3 dimGrid(bx, by);
    dim3 dimBlock(BLOCK_X_2D, BLOCK_Y_2D);

    gm_morph_max_kernel<<<dimGrid, dimBlock>>>(NX, NY, NC, d_image, d_gm);

    // data transfer GPU to CPU
    hipMemcpy(h_gm, d_gm, NX * NY * sizeof(std::uint16_t), hipMemcpyDeviceToHost);
    hipFree(d_image);
    hipFree(d_gm);
}

